#include "hip/hip_runtime.h"
#include "utilities/stopwatch.h"
#include "logging/logging.h"

#include "cuda_ide.h"

struct Event{
    Event(){
        hipEventCreate(&event);
    }

    ~Event(){
        hipEventDestroy(event);
    }

    operator hipEvent_t& (){
        return event;
    }

    static float time(hipEvent_t& start, hipEvent_t& end){
        float f = 0;
        hipEventElapsedTime(&f, start, end);
        return f;
    }

    void record(){
        hipEventRecord(event, nullptr);
    }

    void wait(){
        hipEventSynchronize(event);
    }

    hipEvent_t event;
};

// row-major order
struct MatrixView{
    int width;
    int height;
    int stride;
    float* elements;

    __host__ __device__ float& operator() (int row, int col){
        return elements[row * stride + col];
    }

    __host__ __device__ MatrixView sub_block(int row, int col, int block_size){
        MatrixView sub;
        sub.width = block_size;
        sub.height = block_size;
        sub.stride = stride;
        sub.elements = &(*this)(row * block_size, col * block_size);
        return sub;
    }
};

struct Matrix{
    int width;
    int height;
    int stride;
    float* elements;
    bool cpu;

    Matrix(int rows, int cols, bool cpu=true):
        width(cols), height(cols), stride(width), cpu(cpu)
    {
        std::size_t size = rows * cols * sizeof(float);

        if (cpu){
            elements = static_cast<float*>(malloc(size));
        } else {
           hipMalloc(&elements, size);
        }
    }

    ~Matrix(){
        if (cpu){
            free(elements);
        } else {
            hipFree(elements);
        }
    }

    operator MatrixView(){
        MatrixView v;
        v.width = width;
        v.height = height;
        v.stride = stride;
        v.elements = elements;
        return v;
    }
};



// (n x m) . (m x p) => (n x p)
__global__ void matrix_mult_bad_kernel(float* u, float* v, float* ret, int size) {

    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    const int col = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0;

    if (row < size && col < size){
        for(int i = 0; i < size; ++i) {
            sum += u[i + row * size] + v[col + i * size];
        }

        ret[row * size + col] = sum;
    }
}



template<int tile> __global__
void matrix_mult_tile(MatrixView a, MatrixView b, MatrixView ret, int size) {
    // tile x tile threads running at the same time

    const int block_row = blockIdx.x;
    const int block_col = blockIdx.y;

    // each threads compute one (row, col) pair
    const int row = threadIdx.y;
    const int col = threadIdx.x;
    float sum = 0;

    for (int m = 0; m < (size / tile); ++m) {
        MatrixView sub_a = a.sub_block(block_row, m, tile);
        MatrixView sub_b = b.sub_block(m, block_col, tile);

        // Here each thread is going to fetch 2 values (a_ij b_jk)
        // and store them in the shared location
        // so everybody get access to the data without having to fetch
        // into global memory
        __shared__ float as[tile][tile];
        __shared__ float bs[tile][tile];

        as[row][col] = sub_a(row, col);
        bs[row][col] = sub_a(row, col);

        __syncthreads();

        // All data was fetched we can start computing;
        #pragma unroll
        for(int i = 0; i < tile; ++i){
            sum += as[row][i] * bs[i][col];
        }

        // make sure shared data is not needed anymore
        __syncthreads();
    }

    MatrixView sub_c = ret.sub_block(block_row, block_col, tile);
    sub_c(row, col) = sum;
}





void test_gemm_cuda(int size){
    // [I] [16-02-2020 15:38:07.653] [26786] src/math/gemm_cuda.cu:106 test_gemm_cuda_bad - Total: 2905.5

    Matrix da(size, size, false);
    Matrix db(size, size, false);
    Matrix dc(size, size, false);

    float average = 0;

    for(int j = 0; j < 5; ++j){
        lython::StopWatch<> chrono;
        Event start;
        Event stop;

        start.record();

        for(int i = 0; i < 2; ++i){
            // hipConfigureCall(dim3 gridDim, dim3 blockDim, size_t sharedMem, hipStream_t stream
            // cudaSetupArgumentconst (void* arg, size_t size, size_t offset)
            // hipError_t hipLaunchByPtr (const char *entry)
            // kernel<<<grid, threads, num_extern_shared_bytes, stream>>>(args)

            int tile = 32;

            dim3 dimBlock(tile, tile);
            dim3 dimGrid(size / dimBlock.x, size / dimBlock.y);

            matrix_mult_tile<32><<<dimGrid, dimBlock>>>(da, db, dc, size);
        }

        stop.record();
        stop.wait();
        average += float(chrono.stop());
    }

    kwinfo("Total: {}", average / 10);
}
