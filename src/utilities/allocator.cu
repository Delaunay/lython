#if __has_include(<hip/hip_runtime_api.h>)
#    include <hip/hip_runtime_api.h>

#    include "allocator.h"

namespace lython {
namespace device {

// CUDA alloc is guaranteed to be 256 aligned
void* CUDA::malloc(std::size_t n) {
    void* ptr = nullptr;
    hipMalloc(&ptr, sizeof(float) * n);
    return ptr;
}

bool CUDA::free(void* ptr, std::size_t) {
    hipFree(ptr);
    return true;
}

}  // namespace device
}  // namespace lython
#endif
