#include "hip/hip_runtime.h"
// #include "utilities/pool.h"

#include "logging/logging.h"

#include <iostream>
// #include <cuda/hip/hip_runtime.h>

__global__ void add(float* a, float* b, float* c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n)
        c[index] = a[index] + b[index];
}

int main() {
    // force signal handler to be installed
    info("");

    int    size = 10;
    float *da, *db, *dc;
    float *a, *b, *c;

    a = (float*)malloc(sizeof(float) * size);
    b = (float*)malloc(sizeof(float) * size);
    c = (float*)malloc(sizeof(float) * size);

    for (int i = 0; i < size; ++i) {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
    }

    hipMalloc(&da, sizeof(float) * size);
    hipMalloc(&db, sizeof(float) * size);
    hipMalloc(&dc, sizeof(float) * size);
    // -------------------------------------

    hipMemcpy(da, a, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(float) * size, hipMemcpyHostToDevice);

    int THREADS_PER_BLOCK = 8;
#define TPB THREADS_PER_BLOCK
    add<<<(size + TPB + 1) / TPB, TPB>>>(da, db, dc, size);

    hipMemcpy(c, dc, sizeof(float) * size, hipMemcpyDeviceToHost);

    for (int i = 0; i < size; ++i) {
        std::cout << c[i] << std::endl;
    }

    hipFree(da);
    free(a);
    hipFree(db);
    free(b);
    hipFree(dc);
    free(c);
    return 0;
}
