#include "hip/hip_runtime.h"
// #include "utilities/pool.h"

#include "logging/logging.h"

#include <iostream>
// #include <cuda/hip/hip_runtime.h>


__global__ void add(float*a, float*b, float*c){
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}


int main(){
    // force signal handler to be installed
    info("");

    int size = 10;
    float* da, *db, *dc;
    float* a, *b, *c;

    a = (float*) malloc(sizeof(float) * size);
    b = (float*) malloc(sizeof(float) * size);
    c = (float*) malloc(sizeof(float) * size);

    for(int i = 0; i < size; ++i){
        a[i] = i;
        b[i] = i;
        c[i] = 0;
    }

    hipMalloc(&da, sizeof(float) * size);
    hipMalloc(&db, sizeof(float) * size);
    hipMalloc(&dc, sizeof(float) * size);
    // -------------------------------------

    hipMemcpy(da, a, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(float) * size, hipMemcpyHostToDevice);

    add<<<size,1>>>(da, db, dc);

    hipMemcpy(c, dc, sizeof(float) *size, hipMemcpyDeviceToHost);

    for(int i = 0; i < size; ++i){
        std::cout << c[i] << std::endl;
    }

    hipFree(da); free(a);
    hipFree(db); free(b);
    hipFree(dc); free(c);
    return 0;
}
